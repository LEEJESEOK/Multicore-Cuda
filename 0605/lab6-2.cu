#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "DS_timer.h"

#define f(x) ((x) * (x))


__global__ void trap_kernel(float a, float b, float h, int n, float * sum)
{
	int tID = blockIdx.x * blockDim.x + threadIdx.x;

	if(tID >= n - 1) return;
	
	float x_i = a + h * tID;
	float x_j = a + h * (tID + 1);
	float d = (f(x_i) + f(x_j)) / 2.0;

	atomicAdd(sum, d * h);
}

__global__ void trap_kernel_s1(float a, float b, float h, int n, float * sum)
{
	int tID = blockIdx.x * blockDim.x + threadIdx.x;

	if(tID >= n - 1) return;
	
	__shared__ float localSum;

	float x_i = a + h * tID;
	float x_j = a + h * (tID + 1);
	float d = (f(x_i) + f(x_j)) / 2.0;

	atomicAdd(&localSum, d * h);
	__syncthreads();

	if(threadIdx.x == 0) atomicAdd(sum, localSum);
}

__global__ void trap_kernel_s2(float a, float b, float h, int n, float * sum)
{
	int tID = blockIdx.x * blockDim.x + threadIdx.x;

	if(tID >= n - 1) return;
	
	__shared__ float localVal[64];
	localVal[blockDim.x] = 0;

	float x_i = a + h * tID;
	float x_j = a + h * (tID + 1);
	float d = (f(x_i) + f(x_j)) / 2.0;

	localVal[blockDim.x] = d * h;
	__syncthreads();

	if(threadIdx.x == 0)
	{
		for(int i = 1; i < blockDim.x; i++)
			localVal[0] += localVal[i];

		atomicAdd(sum, localVal[0]);
	}

}

__global__ void trap_kernel_s3(float a, float b, float h, int n, float * sum)
{
	int tID = blockIdx.x * blockDim.x + threadIdx.x;

	if(tID >= n - 1) return;
	
	__shared__ float localVal[64];
	localVal[blockDim.x] = 0;

	float x_i = a + h * tID;
	float x_j = a + h * (tID + 1);
	float d = (f(x_i) + f(x_j)) / 2.0;

	localVal[blockDim.x] = d * h;
	__syncthreads();

	// reduction
	int offset = 1;
	while(offset < blockDim.x)
	{
		if(threadIdx.x % (2 * offset) == 0)
			localVal[threadIdx.x] += localVal[threadIdx.x + offset];

		__syncthreads();
		offset *= 2;
	}

	if(threadIdx.x == 0)
		atomicAdd(sum, localVal[0]);
}

__global__ void trap_kernel_s4(float a, float b, float h, int n, float * sum)
{
	int tID = blockIdx.x * blockDim.x + threadIdx.x;

	if(tID >= n - 1) return;
	
	__shared__ float localVal[64];
	localVal[blockDim.x] = 0;

	float x_i = a + h * tID;
	float x_j = a + h * (tID + 1);
	float d = (f(x_i) + f(x_j)) / 2.0;

	localVal[blockDim.x] = d * h;
	__syncthreads();

	// reduction
	int offset = blockDim.x / 2;
	
	while(offset > 0)
	{
		if(threadIdx.x < offset)
			localVal[threadIdx.x] += localVal[threadIdx.x + offset];

		offset /= 2;

		__syncthreads();
	}

	if(threadIdx.x == 0)
		atomicAdd(sum, localVal[0]);


}

int main()
{
	float a, b, h;
	int n;
   	float sum = 0, * cuda_sum, * d_sum;

	DS_timer timer(6);
	timer.initTimers();

	printf("a > ");
	scanf("%f", &a);
	printf("b > ");
	scanf("%f", &b);
	printf("n > ");
	scanf("%d", &n);

	h = (b - a) / (float) n;

	// CPU version
	timer.setTimerName(0, (char *)"CPU");
	timer.onTimer(0);
	for(int i = 0; i < n - 1; i++)
	{
		float x_i = a + h * i;
		float x_j = a + h * (i + 1);
		float d = (f(x_i) + f(x_j)) / 2.0;
		sum += d * h;
	}
	timer.offTimer(0);
	printf("\tCPU sum : %f\n", sum);

	// CUDA version
	cuda_sum = (float *)malloc(sizeof(float));
	hipMalloc((void **)&d_sum, sizeof(float));
	hipMemset(d_sum, 0, sizeof(float));

	dim3 dimGrid(n / 64, 1, 1);
	dim3 dimBlock(64, 1, 1);

	// Global Sync
	timer.setTimerName(1, (char *)"Global Sync");
	
	timer.onTimer(1);
	trap_kernel<<<dimGrid, dimBlock>>>(a, b, h, n, d_sum);
	hipDeviceSynchronize();
	timer.offTimer(1);
	hipMemcpy(cuda_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
	printf("\tCUDA sum : %f\n", *cuda_sum);	


	// Shared ver1
	hipMemset(d_sum, 0, sizeof(float));
	
	timer.setTimerName(2, (char *)"Shared Ver1");
	timer.onTimer(2);
	trap_kernel_s1<<<dimGrid, dimBlock>>>(a, b, h, n, d_sum);
	hipDeviceSynchronize();
	timer.offTimer(2);
	hipMemcpy(cuda_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
	printf("\tShared v1 sum : %f\n", *cuda_sum);	



	// Shared ver2
	hipMemset(d_sum, 0, sizeof(float));
	
	timer.setTimerName(3, (char *)"Shared Ver2");
	timer.onTimer(3);
	trap_kernel_s2<<<dimGrid, dimBlock>>>(a, b, h, n, d_sum);
	hipDeviceSynchronize();
	timer.offTimer(3);
	hipMemcpy(cuda_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
	printf("\tShared v2 sum : %f\n", *cuda_sum);	



	// Reduction 1
	hipMemset(d_sum, 0, sizeof(float));
	
	timer.setTimerName(4, (char *)"Reduction Ver1");
	timer.onTimer(4);
	trap_kernel_s3<<<dimGrid, dimBlock>>>(a, b, h, n, d_sum);
	hipDeviceSynchronize();
	timer.offTimer(4);
	hipMemcpy(cuda_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
	printf("\tReduction1 sum : %f\n", *cuda_sum);	



	// Reduction 2
	hipMemset(d_sum, 0, sizeof(float));
	
	timer.setTimerName(5, (char *)"Reduction Ver2");
	timer.onTimer(5);
	trap_kernel_s4<<<dimGrid, dimBlock>>>(a, b, h, n, d_sum);
	hipDeviceSynchronize();
	timer.offTimer(5);
	hipMemcpy(cuda_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
	printf("\tReduction2 sum : %f\n", *cuda_sum);	




	timer.printTimer();

	hipFree(d_sum);
}
