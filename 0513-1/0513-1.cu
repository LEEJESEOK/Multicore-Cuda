#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "DS_timer.h"

#define NUM_DATA (512 * 512 * 256)

__global__ void matAdd(int *a, int *b, int *c)
{
	const int NUM_THREAD_IN_BLOCK = blockDim.x * blockDim.y * blockDim.z;

 	int bID = blockIdx.z * (gridDim.y * gridDim.x * NUM_THREAD_IN_BLOCK) + blockIdx.y * (gridDim.x * NUM_THREAD_IN_BLOCK) + (blockIdx.x * (blockDim.x * blockDim.y * blockDim.z));
	int tID = bID + ((blockDim.y * blockDim.x) * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;
	
	c[tID] = a[tID] + b[tID];
}

int main()
{
	// definition
	int *a, *b, *c, *d;
	int *d_a, *d_b, *d_c;

	bool result;

	DS_timer timer(4);


	// init
	a = (int *)malloc(sizeof(int) * NUM_DATA);
	b = (int *)malloc(sizeof(int) * NUM_DATA);
        c = (int *)malloc(sizeof(int) * NUM_DATA);
        d = (int *)malloc(sizeof(int) * NUM_DATA);

	for(int i = 0; i < NUM_DATA; i++)
	{
                a[i] = rand() % 10;
                b[i] = rand() % 10;
	}
	
	timer.initTimers();
	// end of init


	// serial version
	timer.onTimer(0);
	for(int i = 0; i < NUM_DATA; i++)
		c[i] = a[i] + b[i];
	timer.offTimer(0);	


	// cuda version
        hipMalloc((void **) &d_a, sizeof(int) * NUM_DATA);
        hipMalloc((void **) &d_b, sizeof(int) * NUM_DATA);
        hipMalloc((void **) &d_c, sizeof(int) * NUM_DATA);

	// send input data from host to device
	timer.onTimer(1);
        hipMemcpy(d_a, a, sizeof(int) * NUM_DATA, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, sizeof(int) * NUM_DATA, hipMemcpyHostToDevice);
	timer.offTimer(1);

	// kernel call
	timer.onTimer(2);
	matAdd<<<NUM_DATA / 1024, 1024>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	timer.offTimer(2);

	// send result from device to host
	timer.onTimer(3);
	hipMemcpy(d, d_c, sizeof(int) * NUM_DATA, hipMemcpyDeviceToHost);
	timer.offTimer(3);


	// check sequence
	result = true;
	for(int i = 0; i < NUM_DATA; i++)
	{
		if(c[i] != d[i])
		{
			printf("[%d] The results is not matchhed! (%d, %d)\n", i, c[i], d[i]);
			result = false;
		}
	} 

	if(result)
		printf("GPU works well!\n");

	
	timer.printTimer();


        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        delete[] a; delete[] b; delete[] c; delete[] d;

	return 0;
}
