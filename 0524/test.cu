#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void mat_mul(float *a, float *b, float *c, int M, int N, int K){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;
	if(col > N || row > M)
		return;
	for(int k = 0; k < K; k++){
		sum += a[row*K+k] * b[k*N+col];
	}
	c[row*N+col] = sum;
}

int main(){
	hipEvent_t start, end;
	float gpu_time;
	hipEventCreate(&start);
	hipEventCreate(&end);

	const int M = 2048;
	const int N = 1536;
	const int K = 1024;
	// Matrix name: size
	// Matrix A: M * K
	// Matrix B: K * N
	// Matrix C: M * N
	
	float *a, *b, *c;
	a = (float *)malloc(sizeof(float)*M*K);
	b = (float *)malloc(sizeof(float)*K*N);
	c = (float *)malloc(sizeof(float)*M*N);

	for(int i = 0; i < M*K; i++){
		a[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2);
	}
	for(int i = 0; i < K*N; i++){
		b[i] = (float)(rand() - RAND_MAX/2)/(RAND_MAX/2);
	}

	float *da, *db, *dc;
	hipMalloc((void **)&da, sizeof(float)*M*K);
	hipMalloc((void **)&db, sizeof(float)*K*N);
	hipMalloc((void **)&dc, sizeof(float)*M*N);

	hipMemcpy(da, a, sizeof(float)*M*K, hipMemcpyHostToDevice);
	hipMemcpy(db, b, sizeof(float)*K*N, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	
	dim3 dim_block(8, 8, 1);
	dim3 dim_grid((N+8-1)/8, (M+8-1)/8, 1);
	mat_mul<<<dim_grid, dim_block>>>(da, db, dc, M, N, K);

	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&gpu_time, start, end);

	hipMemcpy(c, dc, sizeof(float)*M*N, hipMemcpyDeviceToHost);

	printf("gpu time = %f\n", gpu_time/1000);
	

	free(a);
	free(b);
	free(c);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

